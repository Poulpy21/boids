#include "hip/hip_runtime.h"
#include "headers.hpp"

#ifdef CUDA_ENABLED

#include "options.hpp"
#include "agentData.hpp"

#include "vector.hpp"

__global__ void computeForces(Real *boidData,
                              Real *meanBoidData, 
                              int *meanBoidWeights, 
                              const int nBoids, 
                              const int nMeanBoids,
                              const struct Options *opt) 
{
    int id = blockIdx.x*blockDim.x + threadIdx.x;
    if (id >= nBoids)
        return;

    // Rebuild AgentData
    AgentData boidList(boidData, nBoids), meanBoidList(meanBoidData, nMeanBoids);

    // Compute "internal forces"
    int countSeparation = 0, countCohesion = 0, countAlignment = 0;
    Vector forceSeparation, forceCohesion, forceAlignment;
    Vector thisBoidPosition = boidList.getPosition(id);
    Vector otherBoidPosition;
    for (int i = 0; i < nBoids; i++) {
        if (i != id) {
            otherBoidPosition = boidList.getPosition(i);
            Real dist = (thisBoidPosition - otherBoidPosition).norm();
            if (dist < opt->rSeparation) {
                forceSeparation -= (thisBoidPosition - otherBoidPosition).normalized();
                ++countSeparation; 
            }
            if (dist < opt->rCohesion) {
                forceCohesion += thisBoidPosition;
                ++countCohesion;
            }
            if (dist < opt->rAlignment) {
                forceAlignment += boidList.getVelocity(i);
                ++countAlignment;
            }
        }
    }

    // Compute "external forces"
    for (int i = 0; i < nMeanBoids; i++) {
        otherBoidPosition = boidList.getPosition(i);
        Real dist = (thisBoidPosition - otherBoidPosition).norm();
        Real weight = meanBoidWeights[i];
        if (dist < opt->rSeparation) {
            forceSeparation -= weight * (thisBoidPosition - otherBoidPosition).normalized();
            countSeparation += weight;
        }
        if (dist < opt->rCohesion) {
            forceCohesion += weight * thisBoidPosition;
            countCohesion += weight;
        }
        if (dist < opt->rAlignment) {
            forceAlignment += weight * boidList.getVelocity(i);
            countAlignment += weight;
        }
    }
    
    // Update direction
    Vector direction( opt->wSeparation * ( countSeparation>0 ? forceSeparation/static_cast<Real>(countSeparation) : forceSeparation) +
                      opt->wCohesion   * ( countCohesion  >0 ? forceCohesion  /static_cast<Real>(countCohesion)   : forceCohesion  ) +
                      opt->wAlignment  * ( countAlignment >0 ? forceAlignment /static_cast<Real>(countAlignment)  : forceAlignment ));
    boidList.setDirection(id, direction);
}

void computeForcesKernel(Real *boidData,
                         Real *meanBoidData, 
                         int *meanBoidWeights, 
                         const int nBoids, 
                         const int nMeanBoids,
                         const struct Options *opt) 
{
    dim3 gridDim(1024,1,1); // TODO: max threads/block in globals.hpp using cudaUtils
    dim3 blockDim(ceil((float)nBoids/1024),1,1); 

    computeForces<<<gridDim,blockDim,0,0>>>(boidData, 
                                            meanBoidData, 
                                            meanBoidWeights, 
                                            nBoids, 
                                            nMeanBoids, 
                                            opt);
    
    hipDeviceSynchronize();
    checkKernelExecution();
}



__global__ void applyForces(Real *boidData, const int nBoids, const struct Options *opt) {

    int id = blockIdx.x*blockDim.x + threadIdx.x;
    if (id >= nBoids)
        return;

    // Rebuild AgentData
    AgentData boidList(boidData, nBoids);

    // Update velocity
    Vector velocity = boidList.getVelocity(id) + boidList.getDirection(id);
    Real speed = velocity.norm();
    velocity = (speed > opt->maxVel ? velocity*opt->maxVel/speed : velocity);
    boidList.setVelocity(id, velocity);

    // Update position
    Vector pos = boidList.getPosition(id) + opt->dt * boidList.getVelocity(id);

    // Make sure the boid stays inside the domain
    pos.x = fmod(pos.x, opt->domainSize);
    pos.y = fmod(pos.y, opt->domainSize);
    pos.z = fmod(pos.z, opt->domainSize);
    boidList.setPosition(id, pos);
}

void applyForcesKernel(Real*boidData, const int nBoids, const struct Options *opt) {
    dim3 gridDim(1024,1,1); // TODO: max threads/block in globals.hpp using cudaUtils
    dim3 blockDim(ceil((float)nBoids/1024),1,1); 

    applyForces<<<gridDim,blockDim,0,0>>>(boidData, nBoids, opt);
    
    hipDeviceSynchronize();
    checkKernelExecution();
}


//FIXME reduce sum
__global__ void computeMeanBoid(Real *boidData, const int nBoids, Vector *meanBoid) {

    int id = blockIdx.x*blockDim.x + threadIdx.x;
    if (id >= nBoids)
        return;

    //Rebuild AgentData
    AgentData boidList(boidData, nBoids);

    /*__shared__ Vector sumVector //= Vector(0,0,0);
    sumVector += boidList.getPosition(i);
    *meanBoid =  sumVector / (nBoids>0 ? static_cast<Real>(nBoids) : 1.0);*/
}

void computeMeanBoidKernel(Real *boidData, const int nBoids, Vector *meanBoid) {
    dim3 gridDim(1024,1,1); // TODO: max threads/block in globals.hpp using cudaUtils
    dim3 blockDim(ceil((float)nBoids/1024),1,1); 

    computeMeanBoid<<<gridDim,blockDim,0,0>>>(boidData, nBoids, meanBoid); 
    
    hipDeviceSynchronize();
    checkKernelExecution();
}

#endif
