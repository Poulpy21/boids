#include "hip/hip_runtime.h"

#ifdef __HIPCC__

#include "headers.hpp"
#include "kernel.hpp"
#include "boidGrid.hpp"
#include "vectorMemoryView.hpp"
#include "boidMemoryView.hpp"
#include "thrustVectorMemoryView.hpp"
#include "thrustBoidMemoryView.hpp"
#include "kernel_utilities.cuh"

#ifdef THRUST_ENABLED

namespace kernel {

    namespace boidgrid {

        template <typename T>
            __launch_bounds__(MAX_THREAD_PER_BLOCK)
            __global__ void computeForces(
                    T                   *const __restrict__ boidData,
                    T            const  *const __restrict__ meanBoidPositionData, 
                    unsigned int const  *const __restrict__ uniqueCellIds,
                    unsigned int const  *const __restrict__ uniqueCellCount, 
                    unsigned int const  *const __restrict__ uniqueCellOffsets,
                    int          const  *const __restrict__ validCells,
                    unsigned int const nAgents, 
                    unsigned int const nUniqueIds,
                    unsigned int const nCells) {

                typedef typename MakeCudaVec<T,3>::type vec3; //either float3 or double3

                const unsigned int boidId = blockIdx.y*65535ul*512ul + blockIdx.x*512ul + threadIdx.x;

                if(boidId >= nAgents)
                    return;

                //Reconstruct memory views 
                BoidMemoryView<T>   const boids(boidData, nAgents);
                ConstVectorMemoryView<T> const meanPos(meanBoidPositionData, nUniqueIds);

                //Get infos
                unsigned int const  myCellId         = boids.id[boidId];
                unsigned int const  validCellOffset  = validCells[myCellId];
                unsigned int const  localAgentsCount = uniqueCellCount[validCellOffset];
                unsigned int const  boidArrayOffset  = uniqueCellOffsets[validCellOffset];

                vec3 myPosition;
                myPosition.x = boids.x[boidId];
                myPosition.y = boids.y[boidId];
                myPosition.z = boids.z[boidId];

                //Compute forces
                unsigned int countSeparation=0u, countCohesion=0u, countAlignment=0u;
                vec3 forceSeparation = {}, forceCohesion = {}, forceAlignment = {};
                vec3 neighborPosition;

                //Compute "internal forces"
                for (unsigned int i = 0; i < localAgentsCount; i++) {
                    unsigned int offset = boidArrayOffset + i;
                    if(offset != boidId) {
                        neighborPosition.x = boids.x[offset];
                        neighborPosition.y = boids.y[offset];
                        neighborPosition.z = boids.z[offset];
                        T dist = distance<T>(myPosition, neighborPosition);

                        if(dist < kernel::rSeparation) {
                            forceSeparation.x -= (myPosition.x - neighborPosition.x)/dist;
                            forceSeparation.y -= (myPosition.y - neighborPosition.y)/dist;
                            forceSeparation.z -= (myPosition.z - neighborPosition.z)/dist;
                            countSeparation++;
                        }
                        if(dist < kernel::rCohesion) {
                            forceCohesion.x += neighborPosition.x;
                            forceCohesion.y += neighborPosition.y;
                            forceCohesion.z += neighborPosition.z;
                            countCohesion++;
                        }
                        if(dist < kernel::rAlignment) {
                            forceAlignment.x += boids.vx[offset];
                            forceAlignment.y += boids.vy[offset];
                            forceAlignment.z += boids.vz[offset];
                            countAlignment++;
                        }
                    }
                }

                //Compute "external forces"
                //TODO TODO TODO 

                //Update forces
                vec3 force = {};

                if(countSeparation > 0) {
                    force.x += kernel::wSeparation*forceSeparation.x/countSeparation;
                    force.y += kernel::wSeparation*forceSeparation.y/countSeparation;
                    force.z += kernel::wSeparation*forceSeparation.z/countSeparation;
                }
                if(countCohesion > 0) {
                    force.x += kernel::wCohesion*forceCohesion.x/countCohesion;
                    force.y += kernel::wCohesion*forceCohesion.y/countCohesion;
                    force.z += kernel::wCohesion*forceCohesion.z/countCohesion;
                }
                if(countCohesion > 0) {
                    force.x += kernel::wAlignment*forceAlignment.x/countAlignment;
                    force.y += kernel::wAlignment*forceAlignment.y/countAlignment;
                    force.z += kernel::wAlignment*forceAlignment.z/countAlignment;
                }

                //Integrate in time
                vec3 myVelocity;
                myVelocity.x = boids.vx[boidId] + force.x;
                myVelocity.y = boids.vy[boidId] + force.y;
                myVelocity.z = boids.vz[boidId] + force.z;

                T speed = kernel::norm<T>(myVelocity);

                if(speed > kernel::maxVelocity) {
                    myVelocity.x *= kernel::maxVelocity/speed;
                    myVelocity.y *= kernel::maxVelocity/speed;
                    myVelocity.z *= kernel::maxVelocity/speed;
                }

                myPosition.x += kernel::dt * myVelocity.x;
                myPosition.y += kernel::dt * myVelocity.y;
                myPosition.z += kernel::dt * myVelocity.z;

                //Handle out of domain // Domain looping
                //TODO TODO TODO

                //Compute new id
                //TODO TODO TODO 

                //Write back data to memory
                boids.x[boidId]  = myPosition.x;
                boids.y[boidId]  = myPosition.y;
                boids.z[boidId]  = myPosition.z;
                boids.vx[boidId] = myVelocity.x;
                boids.vy[boidId] = myVelocity.y;
                boids.vz[boidId] = myVelocity.z;
            }


        template <typename T>
            void computeForcesKernel(
                    T                   *const __restrict__ boidData,
                    T            const  *const __restrict__ meanBoidPositionData, 
                    unsigned int const  *const __restrict__ uniqueCellIds,
                    unsigned int const  *const __restrict__ uniqueCellCount, 
                    unsigned int const  *const __restrict__ uniqueCellOffsets,
                    int          const  *const __restrict__ validCells,
                    unsigned int const nAgents, 
                    unsigned int const nUniqueIds,
                    unsigned int const nCells) {

                float nAgents_f = nAgents;

                dim3 dimBlock(MAX_THREAD_PER_BLOCK);
                dim3 dimGrid((unsigned int)ceil(nAgents_f/MAX_THREAD_PER_BLOCK) % 65535, ceil(nAgents_f/(MAX_THREAD_PER_BLOCK*65535.0f)));

                log4cpp::log_console->infoStream() << "[KERNEL::BoidGrid::computeForces] <<<" 
                    << utils::toStringDim(dimBlock) << ", " 
                    << utils::toStringDim(dimGrid)
                    << ">>>";

                computeForces<T><<<dimGrid,dimBlock>>>(
                        boidData,
                        meanBoidPositionData, 
                        uniqueCellIds,
                        uniqueCellCount, 
                        uniqueCellOffsets,
                        validCells,
                        nAgents, 
                        nUniqueIds,
                        nCells);

                CHECK_KERNEL_EXECUTION();
            }



        //__global__ void applyForces(Real *boidData, const int nBoids, const struct Options *opt) {

        //int id = blockIdx.x*blockDim.x + threadIdx.x;
        //if (id >= nBoids)
        //return;

        //Rebuild AgentData
        //AgentData boidList(boidData, nBoids);

        //Update velocity
        //Vector velocity = boidList.getVelocity(id) + boidList.getDirection(id);
        //Real speed = velocity.norm();
        //velocity = (speed > opt->maxVel ? velocity*opt->maxVel/speed : velocity);
        //boidList.setVelocity(id, velocity);

        //Update position
        //Vector pos = boidList.getPosition(id) + opt->dt * boidList.getVelocity(id);

        //Make sure the boid stays inside the domain
        //Real modX = fmod(pos.x, opt->domainSize);
        //Real modY = fmod(pos.y, opt->domainSize);
        //Real modZ = fmod(pos.z, opt->domainSize);
        //pos.x = modX > 0 ? modX : modX + opt->domainSize;
        //pos.y = modY > 0 ? modY : modY + opt->domainSize;
        //pos.z = modZ > 0 ? modZ : modZ + opt->domainSize;
        //boidList.setPosition(id, pos);
        //}

        //void applyForcesKernel(Real*boidData, const int nBoids, const struct Options *opt) {
        //dim3 gridDim(1024,1,1); // TODO: max threads/block in globals.hpp using cudaUtils
        //dim3 blockDim(ceil((float)nBoids/1024),1,1); 

        //applyForces<<<gridDim,blockDim,0,0>>>(boidData, nBoids, opt);

        //hipDeviceSynchronize();
        //checkKernelExecution();
        //}

    }
}

template <typename T>
struct ComputeCellFunctor
{
    unsigned int width, length, height;
    T xmin, ymin, zmin, xmax, ymax, zmax, radius;

    ComputeCellFunctor(const BoidGrid<T> &boidGrid) :
        width(boidGrid.getWidth()), length(boidGrid.getLength()), height(boidGrid.getHeight()),
        xmin(boidGrid.getLocalDomain().min[0]), ymin(boidGrid.getLocalDomain().min[1]), zmin(boidGrid.getLocalDomain().min[2]),
        xmax(boidGrid.getLocalDomain().max[0]), ymax(boidGrid.getLocalDomain().max[2]), zmax(boidGrid.getLocalDomain().max[2]),
        radius(boidGrid.getMaxRadius()) {
        }

    template <typename Tuple>
        __host__ __device__ void operator()(Tuple t) //X Y Z C
        {
            thrust::get<3>(t) = makeId(
                    static_cast<unsigned int>(floor(relativeX(thrust::get<0>(t)) * width)), 
                    static_cast<unsigned int>(floor(relativeY(thrust::get<1>(t)) * length)), 
                    static_cast<unsigned int>(floor(relativeZ(thrust::get<2>(t)) * height))
                    );
        }

    __host__ __device__ T relativeX(T x) { return (x - xmin)/(xmax - xmin);}
    __host__ __device__ T relativeY(T y) { return (y - ymin)/(ymax - ymin);}
    __host__ __device__ T relativeZ(T z) { return (z - zmin)/(zmax - zmin);}
    __host__ __device__ unsigned int makeId(unsigned int x, unsigned int y, unsigned int z) { return (width*length*z + width*y + x); }
};

/*template <typename T>*/
/*using thrust::device_vector<T>::iterator = deviceIterator;*/
typedef thrust::device_vector<Real>::iterator  deviceIterator_real;
typedef thrust::device_vector<unsigned int>::iterator  deviceIterator_ui;

template <typename T>
__host__ void initBoidGridThrustArrays(BoidGrid<T> &boidGrid) {

    unsigned int nAgents = boidGrid.getTotalLocalAgentCount();
    unsigned int nCells = boidGrid.getCellsCount();

    ThrustBoidMemoryView<T> agents_thrust_d(boidGrid.getBoidDeviceMemoryView());
    thrust::device_vector<unsigned int> cellIds(nAgents);

    //copy  X Y Z  VX VY VZ  data to device
    for(unsigned int i = 0; i < BoidMemoryView<T>::N-1; i++) {
        CHECK_THRUST_ERRORS(thrust::copy(boidGrid.getBoidHostMemoryView()[i], boidGrid.getBoidHostMemoryView()[i] + nAgents, agents_thrust_d[i]));
    }

    //compute cell Id for each boid
    CHECK_THRUST_ERRORS(
    thrust::for_each(
            thrust::make_zip_iterator(thrust::make_tuple(agents_thrust_d.x, agents_thrust_d.y, agents_thrust_d.z, agents_thrust_d.id)),
            thrust::make_zip_iterator(thrust::make_tuple(
                    agents_thrust_d.x + nAgents, 
                    agents_thrust_d.y + nAgents,
                    agents_thrust_d.z + nAgents,
                    agents_thrust_d.id + nAgents)),
            ComputeCellFunctor<T>(boidGrid))
    );

    //find the permutation to sort everyone according to the cellIds
    thrust::device_vector<unsigned int> keys(nAgents);
    CHECK_THRUST_ERRORS(thrust::sequence(keys.begin(), keys.end()));
    CHECK_THRUST_ERRORS(thrust::stable_sort_by_key(agents_thrust_d.id, agents_thrust_d.id + nAgents, keys.begin()));

    //find the cells that contains at least one agent
    //and find coresponding array offsets to be copied from the cells
    thrust::device_vector<unsigned int> uniqueIds(agents_thrust_d.id, agents_thrust_d.id + nAgents);
    thrust::device_vector<unsigned int> offsets(nAgents);
    CHECK_THRUST_ERRORS(thrust::sequence(offsets.begin(), offsets.end()));

    thrust::pair<deviceIterator_ui, deviceIterator_ui> end =
        thrust::unique_by_key(uniqueIds.begin(), uniqueIds.end(), offsets.begin());

    unsigned int nUniqueIds = thrust::distance(uniqueIds.begin(), end.first);
    uniqueIds.resize(nUniqueIds);
    offsets.resize(nUniqueIds);

    //count number of boids per key using computed offsets
    thrust::device_vector<unsigned int> count(nUniqueIds);
    CHECK_THRUST_ERRORS(thrust::transform(offsets.begin()+1, offsets.end(), offsets.begin(), count.begin(), thrust::minus<unsigned int>()));
    count[nUniqueIds-1] = nAgents - offsets[nUniqueIds-1];

    //keep filled cells for neighborlookup
    thrust::device_vector<int> validIds(nCells);
    CHECK_THRUST_ERRORS(thrust::fill(validIds.begin(), validIds.end(), -1));
    CHECK_THRUST_ERRORS(
    thrust::scatter(
            thrust::make_counting_iterator<int>(0), 
            thrust::make_counting_iterator<int>(nUniqueIds),
            uniqueIds.begin(), validIds.begin())
    );

    //sort the boids with precomputed permutation
    thrust::device_vector<T> buffer(BoidMemoryView<T>::N*nAgents);
    BoidMemoryView<T> buf_view(buffer.data().get(), nAgents);
    ThrustBoidMemoryView<T> buffer_view(buf_view);

    for(unsigned int i = 0u; i < BoidMemoryView<T>::N; i++) {
        CHECK_THRUST_ERRORS(
        thrust::copy(
                thrust::make_permutation_iterator(agents_thrust_d[i], keys.begin()),
                thrust::make_permutation_iterator(agents_thrust_d[i], keys.end()),
                buffer_view[i])
        );
        CHECK_THRUST_ERRORS(thrust::copy(buffer_view[i], buffer_view[i]+nAgents, agents_thrust_d[i]));
    }

    //allocate and store additional data
    GPUResource<int> &validIds_d = boidGrid.getDeviceValidIds(); 
    GPUResource<unsigned int> &uniqueIds_d = boidGrid.getDeviceUniqueIds(); 
    GPUResource<unsigned int> &offsets_d = boidGrid.getDeviceOffsets(); 
    GPUResource<unsigned int> &count_d = boidGrid.getDeviceCount(); 

    uniqueIds_d.setSize(nUniqueIds);
    uniqueIds_d.allocate();
    offsets_d.setSize(nUniqueIds);
    offsets_d.allocate();
    count_d.setSize(nUniqueIds);
    count_d.allocate();
    validIds_d.setSize(nCells);
    validIds_d.allocate();

    CHECK_THRUST_ERRORS(thrust::copy(uniqueIds.begin(), uniqueIds.end(), uniqueIds_d.wrap()));
    CHECK_THRUST_ERRORS(thrust::copy(offsets.begin(), offsets.end(), offsets_d.wrap()));
    CHECK_THRUST_ERRORS(thrust::copy(count.begin(), count.end(), count_d.wrap()));
    CHECK_THRUST_ERRORS(thrust::copy(validIds.begin(), validIds.end(), validIds_d.wrap()));

    //DEBUG
    //std::cout << "Boid IDs:\t";
    //for(int i = 0; i < nAgents; i++)
    //std::cout << agents_thrust_d.id[i] << " ";
    //std::cout << std::endl;

    //std::cout << "Unique IDs:\t";
    //for(int i = 0; i < uniqueIds.size(); i++)
    //std::cout << uniqueIds[i] << " ";
    //std::cout << std::endl;

    //std::cout << "Offsets:\t";
    //for(int i = 0; i < uniqueIds.size(); i++)
    //std::cout << offsets[i] << " ";
    //std::cout << std::endl;

    //std::cout << "Count:\t\t";
    //for(int i = 0; i < uniqueIds.size(); i++)
    //std::cout << count[i] << " ";
    //std::cout << std::endl;

    //std::cout << "Valid Ids:\t";
    //for(int i = 0; i < validIds.size(); i++)
    //std::cout << validIds[i] << " ";
    //std::cout << std::endl;
}

template <typename T>
__host__ BoidMemoryView<T> computeThrustStep(BoidGrid<T> &boidGrid) {

    unsigned int nAgents    = boidGrid.getTotalLocalAgentCount();
    unsigned int nUniqueIds = boidGrid.getDeviceUniqueIds().size();
    unsigned int nCells     = boidGrid.getCellsCount();

    GPUResource<T>            &agents_d    = boidGrid.getDeviceBoids(); 
    GPUResource<unsigned int> &count_d     = boidGrid.getDeviceCount(); 
    GPUResource<unsigned int> &offsets_d   = boidGrid.getDeviceOffsets(); 
    GPUResource<unsigned int> &uniqueIds_d = boidGrid.getDeviceUniqueIds(); 
    GPUResource<int>          &validIds_d  = boidGrid.getDeviceValidIds(); 

    ThrustBoidMemoryView<T> agents_thrust_d(boidGrid.getBoidDeviceMemoryView());

    // Compute mean positions (only for filled cells)
    thrust::device_vector<T>  means(3*nUniqueIds);
    ThrustVectorMemoryView<T> means_v(means, nUniqueIds);
    {
        thrust::device_vector<unsigned int> buffKeys(nUniqueIds);

        for (unsigned int i = 0; i < 3u; i++) {
            thrust::reduce_by_key(agents_thrust_d.id, agents_thrust_d.id + nAgents,
                    agents_thrust_d[i], buffKeys.begin(), means_v[i], 
                    thrust::equal_to<unsigned int>(), thrust::plus<float>());

            thrust::transform(means_v[i], means_v[i] + nUniqueIds,
                    count_d.wrap(), means_v[i], thrust::divides<float>());
        }
    }

    //Compute forces
    kernel::boidgrid::computeForcesKernel(agents_d.data(), means_v.data(),
            uniqueIds_d.data(), count_d.data(),
            offsets_d.data(), validIds_d.data(),
            nAgents, nUniqueIds, nCells);

    //check for boids that went outside the domain

    BoidMemoryView<T> outOfDomainBoids;
    return outOfDomainBoids;
}


// full specializations
template __host__ void initBoidGridThrustArrays<float> (BoidGrid<float > &boidGrid);
template __host__ void initBoidGridThrustArrays<double>(BoidGrid<double> &boidGrid);

template __host__ BoidMemoryView<float> computeThrustStep<float>(BoidGrid<float> &boidGrid);
template __host__ BoidMemoryView<double> computeThrustStep<double>(BoidGrid<double> &boidGrid);

#endif

#endif
